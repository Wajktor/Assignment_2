#include "hip/hip_runtime.h"


#include <stdio.h>

#define threadBlocks 1
#define threads 256

__global__ void printKernel()
{
	printf("Hello World!  My threadId is %d \n", threadIdx.x);
}

int main()
{
	hipDeviceSynchronize();	
	printKernel <<< threadBlocks, threads >>>();

	hipError_t cudaerr = hipDeviceSynchronize();
	if (cudaerr != hipSuccess) {
		printf("kernel launch failed with error \"%s\".\n",
			hipGetErrorString(cudaerr));
	} else {
		printf("done");
	}
	return 0;
}